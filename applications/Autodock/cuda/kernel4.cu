#include "hip/hip_runtime.h"
/*

AutoDock-GPU, an OpenCL implementation of AutoDock 4.2 running a Lamarckian Genetic Algorithm
Copyright (C) 2017 TU Darmstadt, Embedded Systems and Applications Group, Germany. All rights reserved.
For some of the code, Copyright (C) 2019 Computational Structural Biology Center, the Scripps Research Institute.

AutoDock is a Trade Mark of the Scripps Research Institute.

This library is free software; you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public
License as published by the Free Software Foundation; either
version 2.1 of the License, or (at your option) any later version.

This library is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public
License along with this library; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA

*/


//#define DEBUG_ENERGY_KERNEL4

__global__ void
__launch_bounds__(NUM_OF_THREADS_PER_BLOCK, 1024 / NUM_OF_THREADS_PER_BLOCK)
gpu_gen_and_eval_newpops_kernel(
                                float* pMem_conformations_current,
                                float* pMem_energies_current,
                                float* pMem_conformations_next,
                                float* pMem_energies_next
                               )
// The GPU global function
{
	__shared__ float  offspring_genotype[ACTUAL_GENOTYPE_LENGTH];
	__shared__ int    parent_candidates [4];
	__shared__ float  candidate_energies[4];
	__shared__ int    parents           [2];
	__shared__ int    covr_point        [2];
	__shared__ float  randnums          [10];
	__shared__ float  sBestEnergy       [32];
	__shared__ int    sBestID           [32];
	__shared__ float3 calc_coords       [MAX_NUM_OF_ATOMS];
	__shared__ float  sFloatAccumulator;
	int run_id;
	int temp_covr_point;
	float energy;
	int bestID;

	// In this case this compute-unit is responsible for elitist selection
	if ((blockIdx.x % cData.dockpars.pop_size) == 0) {
		// Find and copy best member of population to position 0
		if (threadIdx.x < cData.dockpars.pop_size)
		{
			bestID = blockIdx.x + threadIdx.x;
			energy = pMem_energies_current[blockIdx.x + threadIdx.x];
		}
		else
		{
			bestID = -1;
			energy = FLT_MAX;
		}
		
		// Scan through population (we already picked up a blockDim's worth above so skip)
		for (int i = blockIdx.x + blockDim.x + threadIdx.x; i < blockIdx.x + cData.dockpars.pop_size; i += blockDim.x)
		{
			float e = pMem_energies_current[i];
			if (e < energy)
			{
				bestID = i;
				energy = e;
			}
		}
		
		// Reduce to shared memory by warp
		int tgx = threadIdx.x & cData.warpmask;
		WARPMINIMUM2(tgx, energy, bestID);
		int warpID = threadIdx.x >> cData.warpbits;
		if (tgx == 0)
		{
			sBestID[warpID] = bestID;
			sBestEnergy[warpID] = fminf(MAXENERGY, energy);
		}
		__threadfence();
		__syncthreads();
		
		// Perform final reduction in warp 0
		if (warpID == 0)
		{
			int blocks = blockDim.x / 32;
			if (tgx < blocks)
			{
				bestID = sBestID[tgx];
				energy = sBestEnergy[tgx];
			}
			else
			{
				bestID = -1;
				energy = FLT_MAX;
			}
			WARPMINIMUM2(tgx, energy, bestID);
			
			if (tgx == 0)
			{
				pMem_energies_next[blockIdx.x] = energy;
				cData.pMem_evals_of_new_entities[blockIdx.x] = 0;
				sBestID[0] = bestID;
			}
		}
		__threadfence();
		__syncthreads();
		
		// Copy best genome to next generation
		int dOffset = blockIdx.x * GENOTYPE_LENGTH_IN_GLOBMEM;
		int sOffset = sBestID[0] * GENOTYPE_LENGTH_IN_GLOBMEM;
		for (int i = threadIdx.x ; i < cData.dockpars.num_of_genes; i += blockDim.x)
		{
			pMem_conformations_next[dOffset + i] = pMem_conformations_current[sOffset + i];
		}
	}
	else
	{
		// Generating the following random numbers: 
		// [0..3] for parent candidates,
		// [4..5] for binary tournaments, [6] for deciding crossover,
		// [7..8] for crossover points, [9] for local search
		for (uint32_t gene_counter = threadIdx.x;
		              gene_counter < 10;
		              gene_counter += blockDim.x)
		{
			randnums[gene_counter] = gpu_randf(cData.pMem_prng_states);
		}
#if 0
		if ((threadIdx.x == 0) && (blockIdx.x == 1))
		{
			printf("%06d ", blockIdx.x);
			for (int i = 0; i < 10; i++)
				printf("%12.6f ", randnums[i]);
			printf("\n");
		}
#endif
		// Determining run ID
		run_id = blockIdx.x / cData.dockpars.pop_size;
		__threadfence();
		__syncthreads();


		if (threadIdx.x < 4) //it is not ensured that the four candidates will be different...
		{
			parent_candidates[threadIdx.x]  = (int) (cData.dockpars.pop_size*randnums[threadIdx.x]); //using randnums[0..3]
			candidate_energies[threadIdx.x] = pMem_energies_current[run_id*cData.dockpars.pop_size+parent_candidates[threadIdx.x]];
		}
		__threadfence();
		__syncthreads();

		if (threadIdx.x < 2)
		{
			// Notice: dockpars_tournament_rate was scaled down to [0,1] in host
			// to reduce number of operations in device
			if (candidate_energies[2*threadIdx.x] < candidate_energies[2*threadIdx.x+1])
			{
				if (/*100.0f**/randnums[4+threadIdx.x] < cData.dockpars.tournament_rate) { // using randnum[4..5]
					parents[threadIdx.x] = parent_candidates[2*threadIdx.x];
				}
				else {
					parents[threadIdx.x] = parent_candidates[2*threadIdx.x+1];
				}
			}
			else
			{
				if (/*100.0f**/randnums[4+threadIdx.x] < cData.dockpars.tournament_rate) {
					parents[threadIdx.x] = parent_candidates[2*threadIdx.x+1];
				}
				else {
					parents[threadIdx.x] = parent_candidates[2*threadIdx.x];
				}
			}
		}
		__threadfence();
		__syncthreads();

		// Performing crossover
		// Notice: dockpars_crossover_rate was scaled down to [0,1] in host
		// to reduce number of operations in device
		if (/*100.0f**/randnums[6] < cData.dockpars.crossover_rate) // Using randnums[6]
		{
			if (threadIdx.x < 2) {
				// Using randnum[7..8]
				covr_point[threadIdx.x] = (int) ((cData.dockpars.num_of_genes-1)*randnums[7+threadIdx.x]);
			}
			__threadfence();
			__syncthreads();
			
			// covr_point[0] should store the lower crossover-point
			if (threadIdx.x == 0) {
				if (covr_point[1] < covr_point[0]) {
					temp_covr_point = covr_point[1];
					covr_point[1]   = covr_point[0];
					covr_point[0]   = temp_covr_point;
				}
			}

			__threadfence();
			__syncthreads();

			for (uint32_t gene_counter = threadIdx.x;
			              gene_counter < cData.dockpars.num_of_genes;
			              gene_counter+= blockDim.x)
			{
				// Two-point crossover
				if (covr_point[0] != covr_point[1]) 
				{
					if ((gene_counter <= covr_point[0]) || (gene_counter > covr_point[1]))
						offspring_genotype[gene_counter] = pMem_conformations_current[(run_id*cData.dockpars.pop_size+parents[0])*GENOTYPE_LENGTH_IN_GLOBMEM+gene_counter];
					else
						offspring_genotype[gene_counter] = pMem_conformations_current[(run_id*cData.dockpars.pop_size+parents[1])*GENOTYPE_LENGTH_IN_GLOBMEM+gene_counter];
				}
				// Single-point crossover
				else
				{
					if (gene_counter <= covr_point[0])
						offspring_genotype[gene_counter] = pMem_conformations_current[(run_id*cData.dockpars.pop_size+parents[0])*GENOTYPE_LENGTH_IN_GLOBMEM+gene_counter];
					else
						offspring_genotype[gene_counter] = pMem_conformations_current[(run_id*cData.dockpars.pop_size+parents[1])*GENOTYPE_LENGTH_IN_GLOBMEM+gene_counter];
				}
			}
		}
		else //no crossover
		{
			for (uint32_t gene_counter = threadIdx.x;
			              gene_counter < cData.dockpars.num_of_genes;
			              gene_counter+= blockDim.x)
			{
				offspring_genotype[gene_counter] = pMem_conformations_current[(run_id*cData.dockpars.pop_size+parents[0])*GENOTYPE_LENGTH_IN_GLOBMEM + gene_counter];
			}
		} // End of crossover

		__threadfence();
		__syncthreads();

		// Performing mutation
		for (uint32_t gene_counter = threadIdx.x;
		              gene_counter < cData.dockpars.num_of_genes;
		              gene_counter+= blockDim.x)
		{
			// Notice: dockpars_mutation_rate was scaled down to [0,1] in host
			// to reduce number of operations in device
			if (/*100.0f**/gpu_randf(cData.pMem_prng_states) < cData.dockpars.mutation_rate)
			{
				// Translation genes
				if (gene_counter < 3) {
					offspring_genotype[gene_counter] += cData.dockpars.abs_max_dmov*(2.0f*gpu_randf(cData.pMem_prng_states)-1.0f);
				}
				// Orientation and torsion genes
				else {
					offspring_genotype[gene_counter] += cData.dockpars.abs_max_dang*(2.0f*gpu_randf(cData.pMem_prng_states)-1.0f);
					map_angle(offspring_genotype[gene_counter]);
				}

			}
		} // End of mutation

		// Calculating energy of new offspring
		__threadfence();
		__syncthreads();
		
		gpu_calc_energy(
		                offspring_genotype,
		                energy,
		                run_id,
		                calc_coords,
		                &sFloatAccumulator
		               );
		
		if (threadIdx.x == 0) {
			pMem_energies_next[blockIdx.x] = energy;
			cData.pMem_evals_of_new_entities[blockIdx.x] = 1;

			#if defined (DEBUG_ENERGY_KERNEL4)
			printf("%-18s [%-5s]---{%-5s}   [%-10.8f]---{%-10.8f}\n", "-ENERGY-KERNEL4-", "GRIDS", "INTRA", interE, intraE);
			#endif
		}


		// Copying new offspring to next generation
		for (uint32_t gene_counter = threadIdx.x;
		              gene_counter < cData.dockpars.num_of_genes;
		              gene_counter+= blockDim.x)
		{
			pMem_conformations_next[blockIdx.x * GENOTYPE_LENGTH_IN_GLOBMEM + gene_counter] = offspring_genotype[gene_counter];
		}
	}
#if 0
	if ((threadIdx.x == 0) && (blockIdx.x == 0))
	{
		printf("%06d %16.8f ", blockIdx.x, pMem_energies_next[blockIdx.x]);
		for (int i = 0; i < cData.dockpars.num_of_genes; i++)
			printf("%12.6f ", pMem_conformations_next[GENOTYPE_LENGTH_IN_GLOBMEM*blockIdx.x + i]);
	}
#endif
}


void gpu_gen_and_eval_newpops(
                              uint32_t blocks,
                              uint32_t threadsPerBlock,
                              float*   pMem_conformations_current,
                              float*   pMem_energies_current,
                              float*   pMem_conformations_next,
                              float*   pMem_energies_next
                             )
{
	gpu_gen_and_eval_newpops_kernel<<<blocks, threadsPerBlock>>>(pMem_conformations_current, pMem_energies_current, pMem_conformations_next, pMem_energies_next);
	LAUNCHERROR("gpu_gen_and_eval_newpops_kernel");
#if 0
	hipError_t status;
	status = hipDeviceSynchronize();
	RTERROR(status, "gpu_gen_and_eval_newpops_kernel");
	status = hipDeviceReset();
	RTERROR(status, "failed to shut down");
	exit(0);
#endif
}
